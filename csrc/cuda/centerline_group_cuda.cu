#include "hip/hip_runtime.h"
#include "centerline_group_cuda.h"

#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

#define THREADS 256

template <typename scalar_t>
__global__ void
centerline_group_kernel(const scalar_t *__restrict__ x, const scalar_t *__restrict__ y,
              const int64_t *__restrict__ ptr_x,
              const int64_t *__restrict__ ptr_y, 
              int64_t *__restrict__ row, int64_t *__restrict__ col, 
              const int64_t n, const int64_t m, const int64_t offset,
              const int64_t num_examples, const int64_t max_num_neighbors) {

  const int64_t n_y = blockIdx.x * blockDim.x + threadIdx.x;
  if (n_y >= m)
    return;

  int64_t count = 0;
  const int64_t example_idx = get_example_idx(n_y, ptr_y, num_examples);

  for (int64_t n_x = ptr_x[example_idx]; n_x < ptr_x[example_idx + 1]; n_x++) {

    int64_t centerline_idx = x[n_x];
    int64_t is_neighbour = y[n_y * offset + centerline_idx];
    
    if (is_neighbour == 1) {
      row[n_y * max_num_neighbors + count] = n_y;
      col[n_y * max_num_neighbors + count] = n_x;
      count++;
    }

    if (count >= max_num_neighbors)
      break;
  }
}

torch::Tensor centerline_group_cuda(const torch::Tensor x, const torch::Tensor y,
                          torch::optional<torch::Tensor> ptr_x,
                          torch::optional<torch::Tensor> ptr_y,
                          const int64_t max_num_neighbors) {
  CHECK_CUDA(x);
  CHECK_CONTIGUOUS(x);
  CHECK_INPUT(x.dim() == 2);

  CHECK_CUDA(y);
  CHECK_CONTIGUOUS(y);
  CHECK_INPUT(y.dim() == 2);

  hipSetDevice(x.get_device());

  if (ptr_x.has_value()) {
    CHECK_CUDA(ptr_x.value());
    CHECK_INPUT(ptr_x.value().dim() == 1);
  } else
    ptr_x = torch::arange(0, x.size(0) + 1, x.size(0),
                          x.options().dtype(torch::kLong));

  if (ptr_y.has_value()) {
    CHECK_CUDA(ptr_y.value());
    CHECK_INPUT(ptr_y.value().dim() == 1);
  } else
    ptr_y = torch::arange(0, y.size(0) + 1, y.size(0),
                          y.options().dtype(torch::kLong));

  hipSetDevice(x.get_device());

  auto row =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());
  auto col =
      torch::full(y.size(0) * max_num_neighbors, -1, ptr_y.value().options());

  dim3 BLOCKS((y.size(0) + THREADS - 1) / THREADS);

  auto stream = at::cuda::getCurrentCUDAStream();
  auto scalar_type = x.scalar_type();
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, scalar_type, "_", [&] {
    centerline_group_kernel<scalar_t><<<BLOCKS, THREADS, 0, stream>>>(
        x.data_ptr<scalar_t>(), y.data_ptr<scalar_t>(),
        ptr_x.value().data_ptr<int64_t>(), ptr_y.value().data_ptr<int64_t>(),
        row.data_ptr<int64_t>(), col.data_ptr<int64_t>(), x.size(0),
        y.size(0), y.size(1),
        ptr_x.value().numel() - 1, max_num_neighbors);
  });

  auto mask = row != -1;
  return torch::stack({row.masked_select(mask), col.masked_select(mask)}, 0);
}
